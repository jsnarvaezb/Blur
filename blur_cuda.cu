#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <sstream>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace std;
using namespace cv;

__device__ int *iToxy(int, int);
__device__ int xyToi(int, int, int);
__device__ uchar3 promedio_actual(uchar3*, int, int, int, int);
__global__ void cudaBlur(uchar3*, uchar3*, int, int, int, int);
inline int _ConvertSMVer2Cores(int, int);

int main(int n, char* argv[])
{
	int hilos = 0;
	int bloques = 0;
	int hilos_totales;
	int cuda_err = hipSuccess;
	int tam_imagen = 0;
	float tam_kernel = -1;
	Mat image;

	//	Verificacion de los parametros para la correcta ejecucion del programa
	if (n != 4) {
		cout << "blur_cuda <ruta img> <Tamano del kernel> <thread>" << endl;
		return 0;
	}

	stringstream ss;
	ss << argv[3];
	ss >> hilos_totales;

	//Se especifican las caracteristicas de la tarjeta
	hipSetDevice(0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int hilos_max = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);

	//Se hace el calculo de bloques e hilos por bloque
	bloques = (hilos_totales / (hilos_max * 2)) + 1;
	hilos = hilos_totales / bloques;

	//	Determina el tam_kernel del kernel
	ss.clear();
	ss << argv[2];
	ss >> tam_kernel;

	//	Se carga la imagen en host
	image = imread(argv[1], CV_LOAD_IMAGE_COLOR);

	//	Determina el tamaño del bloque de memoria para la imagen
	tam_imagen = image.cols*image.rows * sizeof(uchar3);

	//	Reservar la memoria en device para imagen original
	uchar3 *image_d;
	cuda_err = hipMalloc(&image_d, tam_imagen);

	//	Reservar la memoria en device para imagen de respuesta
	uchar3 *ans_d;
	cuda_err = hipMalloc(&ans_d, tam_imagen);

	//	Copiar imagen original al puntero image en device
	cuda_err = hipMemcpy(image_d, image.data, tam_imagen, hipMemcpyHostToDevice);

	//	llamar proceso de blur paralelo
	cudaBlur <<< bloques, hilos >>> (image_d, ans_d, image.cols, image.rows, hilos*bloques, (int)floor(tam_kernel));

	//	Copia la respuesta del apuntador ans_d a image, desde el device al host
	cuda_err = hipMemcpy(image.data, ans_d, tam_imagen, hipMemcpyDeviceToHost);

	//	Liberar memoria en device
	cuda_err = hipFree(image_d);

	cuda_err = hipFree(ans_d);

	imwrite( "blur.jpg", image );

	return 0;
}

/**
* Recorre los puntos del vector de datos de la imagen haciendo el blur a cada uno de ellos
*/
__global__ void cudaBlur(uchar3 *image, uchar3 *ans, int cols, int rows, int n_hilos, int tam_kernel) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = id; i < cols*rows; i += n_hilos) {
		*(ans + i) = promedio_actual(image, i, rows, cols, tam_kernel);
	}
	return;
}

/**
* Convierte i a una cordenada de la forma (x,y).
* Retorna un apuntador con 2 pociciones reservadas.
* En la primera almacena el valor de x
* En la segunda almacena el valor de y
*/
__device__ int *iToxy(int i, int cols) {
	int *ans;
	ans = (int*)malloc(2 * sizeof(int));
	*ans = i%cols;
	*(ans + 1) = i / cols;
	return ans;
}

/**
* convierte una cordenada (x,y) a un valor i para array
* Retorna un entero con el valor de i
*/
__device__ int xyToi(int x, int y, int cols) {
	return cols*y + x;
}

/**
* Halla la suma promediada de los pixeles vecinos en base a un kernel
* image*			Un apuntador a el vector de datos de la imagen
* pos:			El indice del pixel, el indice en base a un array unidimencional
* rows, cols:	dimenciones de la imagen que se esta procesando
* tam_kernel:		El tamano del kernel para los pixeles vecinos
* Retorna un entero con el valor de i
*/
__device__ uchar3 promedio_actual(uchar3* image, int pos, int rows, int cols, int tam_kernel) {
	float  sum_peso;
	float3 sum = {0,0,0};

	sum_peso = 0;

	int *ptr_aux = iToxy(pos, cols);
	int x = *ptr_aux;
	int y = *(ptr_aux + 1);
	free(ptr_aux);

	for (int k = -tam_kernel; k <= tam_kernel; k++) {
		for (int j = -tam_kernel; j <= tam_kernel; j++) {
			if ((x + k) >= 0 && (x + k) < cols &&
				(y + j) >= 0 && (y + j) < rows) {
				float peso = exp(-(k*k + j*j) / (float)(2 * tam_kernel*tam_kernel)) / (3.141592 * 2 * tam_kernel*tam_kernel);
				sum.x += peso * (*(image + xyToi(x + k, y + j, cols))).x;
				sum.y += peso * (*(image + xyToi(x + k, y + j, cols))).y;
				sum.z += peso * (*(image + xyToi(x + k, y + j, cols))).z;
				sum_peso += peso;
			}
		}
	}

	uchar3 ans;

	ans.x = (uchar)floor(sum.x / sum_peso);
	ans.y = (uchar)floor(sum.y / sum_peso);
	ans.z = (uchar)floor(sum.z / sum_peso);

	return ans;
}

/**
 * Funcion de "cuda_helper.h" localizada en samples/common para determinar el numero de cores por multiprocesador del device
 */
inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct
	{
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] =
	{
		{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
		{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
		{ 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class
		{ 0x32, 192 }, // Kepler Generation (SM 3.2) GK10x class
		{ 0x35, 192 }, // Kepler Generation (SM 3.5) GK11x class
		{ 0x37, 192 }, // Kepler Generation (SM 3.7) GK21x class
		{ 0x50, 128 }, // Maxwell Generation (SM 5.0) GM10x class
		{ 0x52, 128 }, // Maxwell Generation (SM 5.2) GM20x class
		{ 0x53, 128 }, // Maxwell Generation (SM 5.3) GM20x class
		{ 0x60, 64 }, // Pascal Generation (SM 6.0) GP100 class
		{ 0x61, 128 }, // Pascal Generation (SM 6.1) GP10x class
		{ 0x62, 128 }, // Pascal Generation (SM 6.2) GP10x class
		{ -1, -1 }
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1)
	{
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
		{
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	return nGpuArchCoresPerSM[index - 1].Cores;
}
